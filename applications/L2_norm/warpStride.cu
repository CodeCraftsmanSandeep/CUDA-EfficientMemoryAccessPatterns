#include <hip/hip_runtime.h>

constexpr unsigned int min_work_per_thread  = 16;

constexpr unsigned int block_size_256_power = 8;
constexpr unsigned int block_size_256       = (1 << block_size_256_power);

constexpr unsigned int block_size_512_power = 9;
constexpr unsigned int block_size_512       = (1 << block_size_512_power);

constexpr unsigned int num_blocks_256_power = 8;
constexpr unsigned int num_blocks_256       = (1 << num_blocks_256_power);

constexpr unsigned int warp_size_power      = 5;
constexpr unsigned int warp_size            = (1 << warp_size_power);

// Custom atomicAdd for datatype: (long long int)
__device__ long long int atomicAdd(long long int* address, long long int val){
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do{
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, assumed + (unsigned long long)val);
    } while (assumed != old);

    return old;
}

template <const unsigned int block_size_power>
__global__ void warpStrideKernel(const unsigned int N, const int* __restrict__ C, const int* __restrict__ B, long long int* __restrict__ intermediary_res)
{
    const unsigned int global_thread_id    =    ((blockIdx.x << block_size_power) + threadIdx.x);
    const unsigned int global_warp_id      =    (global_thread_id >> warp_size_power);

    const unsigned int num_chuncks         =    (N >> warp_size_power);
    const unsigned int num_warps           =    (gridDim.x << (block_size_power - warp_size_power));
    const unsigned int chuncks_per_warp    =    (num_chuncks / num_warps);
    const unsigned int work_per_warp       =    chuncks_per_warp << warp_size_power;

    unsigned int iter                      =    ((work_per_warp * global_warp_id) + (threadIdx.x & 31));
    const unsigned int end                 =    (iter + work_per_warp);
    constexpr unsigned int warp_stride     =    warp_size;
    
    long long int partial_sum = 0;
    while(__builtin_expect(iter < end, 1))
    {
        int diff = C[iter] - B[iter];
        partial_sum += diff * diff;
        iter += warp_stride;
    }

    // Remaining work-items
    iter = work_per_warp * num_warps + global_thread_id;
    if(iter < N)
    {
        int diff = C[iter] - B[iter];
        partial_sum += diff * diff;
    }

    // Tree reduction
    partial_sum += __shfl_down_sync(0xFFFFFFFF, partial_sum, 16);                                               
    partial_sum += __shfl_down_sync(0x0000FFFF, partial_sum,  8);                                               
    partial_sum += __shfl_down_sync(0x000000FF, partial_sum,  4);                                               
    partial_sum += __shfl_down_sync(0x0000000F, partial_sum,  2);                                               
    partial_sum += __shfl_down_sync(0x00000003, partial_sum,  1);                                               
                                                                                                                
    extern __shared__ long long int warp_partial_sum[];                                                         
    const uint8_t lane_id = (threadIdx.x & 31);                                                                
    const uint8_t warp_id = (threadIdx.x >> 5);                                                                
                                                                                                                
    if (lane_id == 0) warp_partial_sum[warp_id] = partial_sum;                                                  
    __syncthreads();                                                                                            
                                                                                                                
    if (warp_id == 0){                                                                                         
        partial_sum = warp_partial_sum[lane_id];                                                               
                                                                                                                
        if constexpr (block_size_power >= 10)  partial_sum += __shfl_down_sync(0xFFFFFFFF, partial_sum, 16);    
        if constexpr (block_size_power >=  9)  partial_sum += __shfl_down_sync(0x0000FFFF, partial_sum,  8);    
        if constexpr (block_size_power >=  8)  partial_sum += __shfl_down_sync(0x000000FF, partial_sum,  4);    
        if constexpr (block_size_power >=  7)  partial_sum += __shfl_down_sync(0x0000000F, partial_sum,  2);    
        if constexpr (block_size_power >=  6)  partial_sum += __shfl_down_sync(0x00000003, partial_sum,  1);    
                                                                                                                
        if (lane_id == 0) atomicAdd(intermediary_res, partial_sum);                                                    
    }
}

__global__ void sqrtKernel (float* __restrict__ L2_norm, const long long int* __restrict__ intermediary_res)
{
    *L2_norm = sqrt((double)*intermediary_res);
}

void compute(const unsigned int N, const int* __restrict__ d_C, const int* __restrict__ d_B, long long int* __restrict__ d_temp_storage, float* __restrict__ d_L2_norm)
{
    hipMemset(d_temp_storage, 0LL, sizeof(long long int));
    hipMemset(d_L2_norm, 0.0f, sizeof(float));

    // Kernel invocation
    if (N < block_size_256 * min_work_per_thread)
    {
        warpStrideKernel <block_size_256_power> 
                <<< 1, block_size_256, warp_size * sizeof(long long int) >>> (N, d_C, d_B, d_temp_storage);
    }
    else if (N < num_blocks_256 * block_size_256 * min_work_per_thread) 
    {
         warpStrideKernel <block_size_256_power> 
                <<< (N >> block_size_256_power) / min_work_per_thread, block_size_256, warp_size * sizeof(long long int) >>> (N, d_C, d_B, d_temp_storage); 
    }
    else if (N < num_blocks_256 * block_size_512 * min_work_per_thread) 
    {
        warpStrideKernel <block_size_256_power> 
                <<< num_blocks_256, block_size_256, warp_size * sizeof(long long int) >>> (N, d_C, d_B, d_temp_storage); 
    }
    else
    {
        warpStrideKernel <block_size_512_power> 
                <<< num_blocks_256, block_size_512, warp_size * sizeof(long long int) >>> (N, d_C, d_B, d_temp_storage);
    }

    sqrtKernel <<< 1, 1 >>> (d_L2_norm, d_temp_storage);
    hipDeviceSynchronize();

    return;
}
