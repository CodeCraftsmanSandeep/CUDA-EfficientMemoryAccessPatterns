#include <hip/hip_runtime.h>

constexpr unsigned int min_work_per_thread  = 16;

constexpr unsigned int block_size_256_power = 8;
constexpr unsigned int block_size_256       = (1 << block_size_256_power);

constexpr unsigned int block_size_512_power = 9;
constexpr unsigned int block_size_512       = (1 << block_size_512_power);

constexpr unsigned int num_blocks_256_power = 8;
constexpr unsigned int num_blocks_256       = (1 << num_blocks_256_power);

constexpr unsigned int warp_size_power      = 5;
constexpr unsigned int warp_size            = (1 << warp_size_power);

// Reduction kernel
template <const unsigned int block_size_power>
__global__ void blockStrideKernel (const unsigned int N, const int* __restrict__ C, const int* __restrict__ B, int* __restrict__ Linfinity_norm)
{
    const unsigned int global_thread_id     =   (threadIdx.x + (blockIdx.x << block_size_power));
    const unsigned int num_chuncks          =   (N >> block_size_power);
    const unsigned int chuncks_per_block    =   (num_chuncks / gridDim.x);

    const unsigned int work_per_block       =   (chuncks_per_block << block_size_power);
    unsigned int iter                       =   ((work_per_block * blockIdx.x) + threadIdx.x);
    const unsigned int end                  =   (iter + work_per_block);
    constexpr unsigned int block_stride     =   (1 << block_size_power);

    // Block-stride access
    int partial_sum = 0;
    while(__builtin_expect(iter < end, 1)){
        partial_sum = max(partial_sum, abs(C[iter] - B[iter]));
        iter += block_stride;        
    }

    // Remaining work-items
    iter = ((work_per_block * gridDim.x) + global_thread_id);
    if(iter < N) partial_sum = max(partial_sum, abs(C[iter] - B[iter]));

    // Tree reduction
    partial_sum = max(partial_sum, __shfl_down_sync(0xFFFFFFFF, partial_sum, 16));
    partial_sum = max(partial_sum, __shfl_down_sync(0x0000FFFF, partial_sum,  8));
    partial_sum = max(partial_sum, __shfl_down_sync(0x000000FF, partial_sum,  4));
    partial_sum = max(partial_sum, __shfl_down_sync(0x0000000F, partial_sum,  2));
    partial_sum = max(partial_sum, __shfl_down_sync(0x00000003, partial_sum,  1));

    extern __shared__ int warp_partial_sum[];
    const uint8_t lane_id = (threadIdx.x & 31);
    const uint8_t warp_id = (threadIdx.x >> 5);

    if (lane_id == 0) warp_partial_sum[warp_id] = partial_sum;
    __syncthreads();

    if (warp_id == 0){
        partial_sum = warp_partial_sum[lane_id];

        if constexpr (block_size_power >= 10)  partial_sum = max(partial_sum, __shfl_down_sync(0xFFFFFFFF, partial_sum, 16));
        if constexpr (block_size_power >=  9)  partial_sum = max(partial_sum, __shfl_down_sync(0x0000FFFF, partial_sum,  8));
        if constexpr (block_size_power >=  8)  partial_sum = max(partial_sum, __shfl_down_sync(0x000000FF, partial_sum,  4));
        if constexpr (block_size_power >=  7)  partial_sum = max(partial_sum, __shfl_down_sync(0x0000000F, partial_sum,  2));
        if constexpr (block_size_power >=  6)  partial_sum = max(partial_sum, __shfl_down_sync(0x00000003, partial_sum,  1));

        if (lane_id == 0) atomicMax(Linfinity_norm, partial_sum);
    }
}

void compute(const unsigned int N, const int* __restrict__ d_C, const int* __restrict__ d_B, int* __restrict__ d_Linfinity_norm)
{
    hipMemset(d_Linfinity_norm, 0, sizeof(int));

    // Kernel invocation
    if (N < block_size_256 * min_work_per_thread)
    {
        blockStrideKernel       <block_size_256_power> 
                <<< 1, block_size_256, warp_size * sizeof(int) >>> (N, d_C, d_B, d_Linfinity_norm);
    }
    else if (N < num_blocks_256 * block_size_256 * min_work_per_thread) 
    {
        blockStrideKernel       <block_size_256_power>
                <<< (N >> block_size_256_power) / min_work_per_thread, block_size_256, warp_size * sizeof(int) >>> (N, d_C, d_B, d_Linfinity_norm);
    }
    else if (N < num_blocks_256 * block_size_512 * min_work_per_thread) 
    {
        blockStrideKernel       <block_size_256_power> 
               <<< num_blocks_256, block_size_256, warp_size * sizeof(int) >>> (N, d_C, d_B, d_Linfinity_norm);
    }
    else
    {
        blockStrideKernel       <block_size_512_power> 
              <<< num_blocks_256, block_size_512, warp_size * sizeof(int) >>> (N, d_C, d_B, d_Linfinity_norm);
    }

    hipDeviceSynchronize();
}
